
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#define MAX_LENGTH 614400

// nsys profile -t nvtx,cuda --stats=true --force-overwrite true --wait=all -o my_report ./my_app

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

void sequential_histogram(char *data, unsigned int *histogram, int length)
{
    for (int i = 0; i < length; i++)
    {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) // check if we have an alphabet char
            histogram[alphabet_position / 6]++;               // we group the letters into blocks of 6
    }
}

__global__ void histogram_kernel(char *data, unsigned int *histogram, int length)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int section_size = (length - 1) / (blockDim.x * gridDim.x) + 1;
    int start = i * section_size;
    // All threads handle blockDim.x * gridDim.x
    // consecutive elements
    for (size_t k = 0; k < section_size; k++)
    {
        if (start + k < length)
        {
            int alphabet_position = data[start + k] - 'a';
            if (alphabet_position >= 0 && alphabet_position < 26)
                atomicAdd(&(histogram[alphabet_position / 6]), 1);
        }
    }
}

int main(int argc, char *argv[])
{
    FILE *fp = fopen("test.txt", "read");

    if (argc != 2)
    {
        printf("Usage: ./exec BLOCKDIM\n");
        return 0;
    }

    int BLOCKDIM = atoi(argv[1]);

    // unsigned char text[MAX_LENGTH];
    char *text = (char *)malloc(sizeof(char) * MAX_LENGTH);
    char *text_d;
    size_t len = 0;
    size_t read;
    unsigned int histogram[5] = {0};
    unsigned int histogram_hw[5] = {0};
    unsigned int *histogram_d;
    double start_cpu, end_cpu, start_gpu, end_gpu;

    if (fp == NULL)
        exit(EXIT_FAILURE);

    while ((read = getline(&text, &len, fp)) != -1)
    {
        printf("Retrieved line of length %ld:\n", read);
    }
    fclose(fp);

    start_cpu = get_time();
    sequential_histogram(text, histogram, len);
    end_cpu = get_time();

    CHECK(hipMalloc(&text_d, len * sizeof(char)));                              // allocate space for the input array on the GPU
    CHECK(hipMalloc(&histogram_d, 5 * sizeof(unsigned int)));                   // and for the histogram
    CHECK(hipMemcpy(text_d, text, len * sizeof(char), hipMemcpyHostToDevice)); // copy input data on the gpu

    dim3 blocksPerGrid((len + BLOCKDIM - 1) / BLOCKDIM, 1, 1);
    dim3 threadsPerBlock(BLOCKDIM, 1, 1);
    start_gpu = get_time();
    histogram_kernel<<<blocksPerGrid, threadsPerBlock>>>(text_d, histogram_d, len);
    CHECK_KERNELCALL();

    hipDeviceSynchronize();
    end_gpu = get_time();
    CHECK(hipMemcpy(histogram_hw, histogram_d, 5 * sizeof(unsigned int), hipMemcpyDeviceToHost)); // copy data back from the gpu
    for (size_t i = 0; i < 5; i++)
    {
        if (histogram[i] != histogram_hw[i])
        {
            printf("Error on GPU at index: %ld\n", i);
            return 0;
        }
    }
    printf("ALL GPU OK\n");

    printf("CPU Sort Time: %.5lf\n", end_cpu - start_cpu);
    printf("GPU Sort Time: %.5lf\n", end_gpu - start_gpu);

    CHECK(hipFree(text_d));
    CHECK(hipFree(histogram_d));

    return 1;
}
